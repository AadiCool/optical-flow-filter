#include "hip/hip_runtime.h"
/**
 * \file flowfilter.cu
 * \brief Optical flow filter classes.
 * \copyright 2015, Juan David Adarve, ANU. See AUTHORS for more details
 * \license 3-clause BSD, see LICENSE for more details
 */


#include <iostream>
#include <exception>
#include <cmath>

#include "flowfilter/gpu/util.h"
#include "flowfilter/gpu/error.h"
#include "flowfilter/gpu/flowfilter.h"

namespace flowfilter {
namespace gpu {


FlowFilter::FlowFilter() :
    Stage() {

    __height = 0;
    __width = 0;
    __configured = false;
}

FlowFilter::FlowFilter(const int height, const int width) :
    FlowFilter(height, width, 1, 1.0, 1.0) {
}

FlowFilter::FlowFilter(const int height, const int width,
        const int smoothIterations,
        const float maxflow,
        const float gamma) :
    Stage() {

    if(height <= 0) {
        std::cerr << "ERROR: FlowFilter::FlowFilter(): height should be greater than zero: "
            << height << std::endl;
        throw std::exception();
    }

    if(width <= 0) {
        std::cerr << "ERROR: FlowFilter::FlowFilter(): width should be greater than zero: "
            << width << std::endl;
        throw std::exception();
    }

    __height = height;
    __width = width;
    __configured = false;

    configure();
    setGamma(gamma);
    setMaxFlow(maxflow);
    setSmoothIterations(smoothIterations);
}

FlowFilter::~FlowFilter() {
    // nothing to do
}


void FlowFilter::configure() {

    // connect the blocks
    __inputImage = GPUImage(__height, __width, 1, sizeof(unsigned char));
    __imageModel = ImageModel(__inputImage);

    // dummy flow field use to instanciate the update block
    // This is necessary to break the circular dependency
    // between propagation and update blocks.
    GPUImage dummyFlow(__height, __width, 2, sizeof(float));

    // FIXME: find good default values
    __update = FlowUpdate(dummyFlow,
        __imageModel.getImageConstant(),
        __imageModel.getImageGradient(),
        1.0, 1.0);

    __smoother = FlowSmoother(__update.getUpdatedFlow(), 1);

    __propagator = FlowPropagator(__smoother.getSmoothedFlow(), 1);

    // set the input flow of the update block to the output
    // of the propagator. This replaces dummyFlow previously
    // assigned to the update
    __update.setInputFlow(__propagator.getPropagatedFlow());

    __configured = true;
    __firstLoad = true;
}


void FlowFilter::compute() {

    startTiming();

    if(__firstLoad) {

        // TODO: set old image to input image
        // set old flow to zero

        __firstLoad = false;
    }

    // compute image model
    __imageModel.compute();

    // propagate old flow
    __propagator.compute();

    // update
    __update.compute();

    // smooth updated flow
    __smoother.compute();

    stopTiming();
}

void FlowFilter::loadImage(flowfilter::image_t& image) {
    __inputImage.upload(image);
}

void FlowFilter::downloadFlow(flowfilter::image_t& flow) {
    __update.getUpdatedFlow().download(flow);
}

GPUImage FlowFilter::getFlow() {
    return __update.getUpdatedFlow();
}


float FlowFilter::getGamma() const {
    return __update.getGamma();
}


void FlowFilter::setGamma(const float gamma) {
    __update.setGamma(gamma);
}


float FlowFilter::getMaxFlow() const {
    return __update.getMaxFlow();
}


void FlowFilter::setMaxFlow(const float maxflow) {
    __update.setMaxFlow(maxflow);
    __propagator.setIterations(int(ceilf(maxflow)));
}


int FlowFilter::getSmoothIterations() const {
    return __smoother.getIterations();
}


void FlowFilter::setSmoothIterations(const int N) {
    __smoother.setIterations(N);
}


int FlowFilter::getPropagationIterations() const {
    return __propagator.getIterations();
}


}; // namespace gpu
}; // namespace flowfilter