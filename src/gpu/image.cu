/**
 * \file image.cu
 * \brief type declarations for GPU image buffers.
 * \copyright 2015, Juan David Adarve, ANU. See AUTHORS for more details
 * \license 3-clause BSD, see LICENSE for more details
 */

#include <iostream>

#include "flowfilter/gpu/image.h"
#include "flowfilter/gpu/gpu_deleter.h"
#include "flowfilter/gpu/error.h"

namespace flowfilter {
    namespace gpu {

        //#################################################
        // GPUImage
        //#################################################
        GPUImage::GPUImage() {
            __width = 0;
            __height = 0;
            __depth = 0;
            __pitch = 0;
            __itemSize = 0;
            __ptr_dev = std::shared_ptr<void> {nullptr, gpu_deleter<void>()};
        }

        GPUImage::GPUImage(const int height, const int width,
            const int depth, const int itemSize) {

            __height = height;
            __width = width;
            __depth = depth;
            __itemSize = itemSize;

            // allocate memory in GPU space
            allocate();
        }

        GPUImage::~GPUImage() {

            // nothing to do
            // device buffer is released by gpu_deleter
            std::cout << "GPUImage::~GPUImage(): [" << 
                __height << ", " << __width << ", " << __depth << "]" << std::endl;
        }

        int GPUImage::height() const {
            return __height;
        }

        int GPUImage::width() const {
            return __width;
        }

        int GPUImage::depth() const {
            return __depth;
        }

        int GPUImage::pitch() const {
            return __pitch;
        }

        int GPUImage::itemSize() const {
            return __itemSize;
        }


        void GPUImage::upload(flowfilter::image_t& img) {

            // check if device memory is allocated
            if(!__ptr_dev) {

                // set resolution to input image
                __width = img.width;
                __height = img.height;
                __depth = img.depth;
                __itemSize = img.itemSize;

                // allocate memory
                allocate();
            }

            // compare shapes
            if(compareShape(img)) {

                // print first 5 elements of img
                // for(int i = 0; i < 5; i ++) {
                //     std::cout << i << ": " << static_cast<float*>(img.data)[i] << std::endl;
                // }

                // issue synchronous memory copy
                checkError(hipMemcpy2D(__ptr_dev.get(), __pitch, img.data, img.pitch,
                    __width*__depth*__itemSize, __height,
                    hipMemcpyHostToDevice));

                // TODO: add support for asynchronous copy

            } else {
                std::cerr << "ERROR: GPUImage::upload(): shapes do not match" << std::endl;
                return; // TODO: throw exception
            }
        }

        void GPUImage::download(flowfilter::image_t& img) const {

            if(!__ptr_dev) {
                std::cerr << "ERROR: GPUImage::download(): unallocated image" << std::endl;
                return; // TODO: throw exception
            }

            if(compareShape(img)) {

                // issue synchronous memory copy
                checkError(hipMemcpy2D(img.data, img.pitch, __ptr_dev.get(), __pitch,
                    __width*__depth*__itemSize, __height, hipMemcpyDeviceToHost));

                // print first 5 elements of img
                // for(int i = 0; i < 5; i ++) {
                //     std::cout << i << ": " << static_cast<float*>(img.data)[i] << std::endl;
                // }

            } else {
                std::cerr << "ERROR: GPUImage::download(): shapes do not match" << std::endl;
                return; // TODO: throw exception
            }
        }


        void GPUImage::allocate() {

            std::cout << "GPUImage::allocate()" << std::endl;

            void* buffer_dev = nullptr;
            hipError_t err = hipMallocPitch(&buffer_dev, &__pitch,
                __width*__depth*__itemSize, __height);

            // create a new shared pointer
            __ptr_dev = std::shared_ptr<void> {buffer_dev, gpu_deleter<void>()};

            std::cout << "\tpitch: " << __pitch << std::endl;

            if(err != hipSuccess) {
                std::cerr << "ERROR: GPUImage device memory allocation: "
                    << hipGetErrorString(err) << std::endl;
                // TODO: throw exception?
            }
        }

        bool GPUImage::compareShape(const flowfilter::image_t& img) const {

            return __height == img.height &&
                __width == img.width &&
                __depth == img.depth &&
                __itemSize == img.itemSize;
        }

    }; // namespace gpu
}; // namespace flowfilter

        