/**
 * \file pipeline.cu
 * \brief type declarations vision pipelines.
 * \copyright 2015, Juan David Adarve, ANU. See AUTHORS for more details
 * \license 3-clause BSD, see LICENSE for more details
 */


#include "flowfilter/gpu/pipeline.h"

#include <iostream>

namespace flowfilter {
    namespace gpu {

        //#################################################
        // Stage
        //#################################################
        Stage::Stage() :
            Stage(0) {
        }

        Stage::Stage(hipStream_t stream) {

            __stream = stream;
            __elapsedTime = 0.0f;

            hipError_t startErr = hipEventCreate(&__start);
            hipError_t stopErr = hipEventCreate(&__stop);

            if(startErr != hipSuccess || stopErr != hipSuccess) {
                std::cerr << "Stage::Stage(): error creating CUDA events: "
                << hipGetErrorString(startErr) << " - "
                << hipGetErrorString(stopErr) << std::endl;

                // TODO: throw exception
            }
        }

        Stage::~Stage() {

            hipError_t startErr = hipEventDestroy(__start);
            hipError_t stopErr = hipEventDestroy(__stop);

            if(startErr != hipSuccess || stopErr != hipSuccess) {
                std::cerr << "Stage::Stage(): error destroying CUDA events: "
                << hipGetErrorString(startErr) << " - "
                << hipGetErrorString(stopErr) << std::endl;

                // TODO: throw exception
            }
        }


        void Stage::startTiming() {
            hipEventRecord(__start, __stream);
        }

        void Stage::stopTiming() {
            hipEventRecord(__stop, __stream);
            hipEventSynchronize(__stop);
            hipEventElapsedTime(&__elapsedTime, __start, __stop);
        }

        /**
         * \brief return computation elapsed time in milliseconds
         */
        float Stage::elapsedTime() const {
            return __elapsedTime;
        }


        //#################################################
        // EmptyStage
        //#################################################
        EmptyStage::EmptyStage() :
            Stage() {

            // nothing to do
        }

        EmptyStage::~EmptyStage() {
            // nothing to do
        }

        void EmptyStage::configure() {
            // nothing to do...
        }

        void EmptyStage::compute() {
            
            startTiming();

            // no operation to be performed

            stopTiming();
        }

    };
};